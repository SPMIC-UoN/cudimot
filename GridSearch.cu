#include "hip/hip_runtime.h"
/* GridSearch.cu

   Moises Hernandez-Fernandez - FMRIB Image Analysis Group
   
   Copyright (C) 2005 University of Oxford */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk

    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK


    LICENCE

    FMRIB Software Library, Release 6.0 (c) 2018, The University of
    Oxford (the "Software")

    The Software remains the property of the Oxford University Innovation
    ("the University").

    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.

    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.

    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.

    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    fsl@innovation.ox.ac.uk quoting Reference Project 9564, FSL.*/

#include "GridSearch.h"
#include "functions_gpu.h"
#include "modelparameters.h"
#include "macro_numerical.h"
#include "modelfunctions.h"

namespace Cudimot{

#define VOXELS_BLOCK 8
#define THREADS_VOXEL 32 // Multiple of 32: Threads collaborating to compute a voxel. Do not change this, otherwise Synchronization will be needed

  __constant__ int gridParams [NPARAMS]; // may not use all, but max is NPARAMS
  __constant__ int GSbound_types [NPARAMS];
  __constant__ float GSbounds_min [NPARAMS];
  __constant__ float GSbounds_max [NPARAMS];

  template <typename T>
  __device__ inline bool checkBounds(T* params)
  {
    #pragma unroll
    for(int p=0;p<NPARAMS;p++){
      if(GSbound_types[p]==BMIN){
	// Bounded with only min
	if (params[p] < GSbounds_min[p]) 
	  return false;
      }else if(GSbound_types[p]==BMAX){
	// Bounded with only max
	if (params[p] > GSbounds_max[p])
	   return false;
      }else if(GSbound_types[p]==BMINMAX){
	// Bounded with min & max
	if (params[p] < GSbounds_min[p])
	  return false;
	else if (params[p] > GSbounds_max[p])
	  return false;
      }
    }
    return true;
  }

  template <typename T, bool DEBUG>
  __device__ inline void Cost_Function(int idSubVOX,
				       int nmeas,
				       int CFP_Tsize,
				       T* measurements,
				       T* parameters,
				       T* CFP,
				       T* FixP,
				       double* result,
				       int debugVOX)
  {
    int idMeasurement=idSubVOX;
    T accumulated_error=(T)0.0;
    
    int nmeas2compute = nmeas/THREADS_VOXEL;
    if (idSubVOX<(nmeas%THREADS_VOXEL)) nmeas2compute++;
    
    for(int iter=0;iter<nmeas2compute;iter++){
      T* myCFP = &CFP[idMeasurement*CFP_Tsize];
      T pred_error=Predicted_Signal(NPARAMS,parameters,myCFP,FixP);
      
      if(DEBUG){
	int idVOX= (blockIdx.x*VOXELS_BLOCK)+int(threadIdx.x/THREADS_VOXEL);
	if(idVOX==debugVOX && idSubVOX==0){
	  printf("PredictedSignal[%i]: %f\n",idMeasurement,pred_error);
	}
      }
      
      pred_error=pred_error-measurements[idMeasurement];
      accumulated_error+=pred_error*pred_error;
      idMeasurement+=THREADS_VOXEL;
    }
     
    #pragma unroll
    for(int offset=THREADS_VOXEL/2; offset>0; offset>>=1){
      accumulated_error+= shfl_down(accumulated_error,offset);
    }
    if(idSubVOX==0){
      *result=accumulated_error;
      if(DEBUG){
	int idVOX= (blockIdx.x*VOXELS_BLOCK)+int(threadIdx.x/THREADS_VOXEL);
	if(idVOX==debugVOX && idSubVOX==0){
	   printf("COST FUNTION: %f\n",*result);
	}
      }
    }
  }
  
  template <typename T, bool DEBUG>
  __global__ void gridSearch_kernel(
				    int nGridParams,
				    int gridCombs,
				    int nmeas, // nmeasurements
				    int CFP_Tsize, // common fixed params: size*M-measurements
				    int FixP_Tsize, // fixed params: size*Nvoxels 
				    T* meas, // measurements
				    T* grid, // values to try
				    T* parameters, // model parameters 
				    T* CFP_global, // common fixed model parameters
				    T* FixP, // fixed model parameters
				    int debugVOX)
  {
    // 1 block of threads process several voxels
    // Each warp processes 1 voxel
    int idVOX= (blockIdx.x*VOXELS_BLOCK)+int(threadIdx.x/THREADS_VOXEL);
    int idVOX_inBlock =  threadIdx.x/THREADS_VOXEL;
    int idSubVOX= threadIdx.x%THREADS_VOXEL;
    bool leader = (idSubVOX==0);  // Some steps are performed by only one thread of the warp
    
    ////////// DYNAMIC SHARED MEMORY ///////////
    extern  __shared__ double shared[];				//Size:
    double* pcf = (double*) shared;    				//VOXELS_BLOCK 
    double* ncf = (double*) &pcf[VOXELS_BLOCK];			//VOXELS_BLOCK
    T* CFP = (T*) &ncf[VOXELS_BLOCK];		       		//nmeas*CMP_Tsize
    T* params = (T*) &CFP[nmeas*CFP_Tsize]; 			//NPARAMS*VOXELS_BLOCK
    T* trial_params = (T*) &params[NPARAMS*VOXELS_BLOCK];       //NPARAMS*VOXELS_BLOCK
    int* boundsTest = (int*) &trial_params[NPARAMS*VOXELS_BLOCK]; //VOXELS_BLOCK
    ////////////////////////////////////////////
    
    /// Copy common fixed model parameters to Shared Memory ///
    if(threadIdx.x==0){ // only one thread of the whole block. Common to all voxels
      for(int i=0;i<nmeas*CFP_Tsize;i++){
	CFP[i]=CFP_global[i];
      }
    }
    ///////////////////////////////////////////////////////////
    
    ///////// each voxel/warp of the block points to its data///////////
    meas = &meas[idVOX*nmeas]; //Global memory
    FixP = &FixP[idVOX*FixP_Tsize]; // Global memory
    pcf = &pcf[idVOX_inBlock];
    ncf = &ncf[idVOX_inBlock];
    params = (T*)&params[idVOX_inBlock*NPARAMS];
    trial_params = (T*)&trial_params[idVOX_inBlock*NPARAMS];
    boundsTest = (int*)&boundsTest[idVOX_inBlock];
    
    /// Ititialise shared values of each voxel: only the leader///
    if(leader){
      #pragma unroll
      for(int i=0;i<NPARAMS;i++){
	params[i]=parameters[idVOX*NPARAMS+i];
      }
      if(DEBUG){
	if(idVOX==debugVOX){
	  printf("\n ----- GridSearch GPU algorithm: voxel %i -----\n",idVOX);
	  for(int i=0;i<NPARAMS;i++){
	    printf("Initial Parameter[%i]: %f\n",i,params[i]);
	  }
	  for(int i=0;i<CFP_Tsize;i++){
	    printf("Commonn Fixed Params[%i]: ",i);
	    for(int j=0;j<nmeas;j++){
	      printf("%f ",CFP_global[j*CFP_Tsize+i]);
	    }
	    printf("\n");
	  }
	  printf("Fix Parameters: ");
	  for(int i=0;i< FixP_Tsize;i++){
	    printf("%f, ",FixP[i]);
	  }
	  printf("\n--------------------------------------------------------\n",idVOX);  
	}
      }
    }
    // __threadfence_block();
    __syncthreads();
    ///////////////////////////////////////////
    
    //Cost_Function<T,DEBUG>(idSubVOX,nmeas,CFP_Tsize,meas,params,CFP,FixP,pcf,debugVOX);
    if(leader){
      *pcf=9e20;
    }
    if(DEBUG){
      if(idVOX==debugVOX&&leader){
	printf("--------------------------------------------------------\n");  
      }
    }
    
    for(int comb=0;comb<gridCombs;comb++){
      if(leader){
        #pragma unroll
	for(int i=0; i<NPARAMS; i++){
	  trial_params[i]= params[i];
	}
	for(int i=0;i<nGridParams;i++){
	  trial_params[gridParams[i]]= grid[(comb*nGridParams)+i];
	}
	*boundsTest=checkBounds(trial_params);
      }
    
      if(DEBUG){
	if(idVOX==debugVOX&&leader){
	  printf("---------------------- Combination %i ---------------------\n",comb);
	  printf("Parameters: "); 
	  for(int i=0;i<NPARAMS;i++) printf("%f ",trial_params[i]);
	  printf("\nBoundsTest: %i\n",*boundsTest);
	}
      }
      
      //__threadfence_block();
      __syncthreads();    
      
      if(*boundsTest){
	Cost_Function<T,DEBUG>(idSubVOX,nmeas,CFP_Tsize,meas,trial_params,CFP,FixP,ncf,debugVOX);
	//__threadfence_block(); // Leader may be faster an update params
	__syncthreads(); 
      
	if(leader){
	  if ((*ncf) < (*pcf)){ 
            #pragma unroll
	    for(int i=0;i<NPARAMS;i++){
	      params[i]=trial_params[i];
	    }
	    *pcf=*ncf;
	  }
	}
      }
      if(DEBUG){
	if(leader&&idVOX==debugVOX){
	  printf("--------------------------------------------------------\n");  
	}
      }
      //__threadfence_block();
      __syncthreads(); 
    }
    
    if(leader){
      // save parameters in global
      #pragma unroll
      for(int i=0;i<NPARAMS;i++){
	parameters[idVOX*NPARAMS+i]=params[i];
      }
      if(DEBUG){
	if(idVOX==debugVOX){
	  for(int i=0;i<NPARAMS;i++){
	    printf("Final Parameter[%i]: %f\n",i,params[i]);
	  }
	}
      }
    }
  }
  
  
  template <typename T>
  GridSearch<T>::GridSearch(int nGP, vector<int> gP, int gC, T* grid_host,
			    vector<int> bou_types, vector<T> bou_min, 
			    vector<T> bou_max)
  {
    cudimotOptions& opts = cudimotOptions::getInstance();
    if(opts.gridSearch.value()!=""){
      nGridParams=nGP;
      gridParams_host=new int[NPARAMS];
      for(int i=0;i<NPARAMS;i++) gridParams_host[i]=0;
      for(int i=0;i<gP.size();i++) gridParams_host[i]=gP[i];
      hipMemcpyToSymbol(HIP_SYMBOL(gridParams),gridParams_host,NPARAMS*sizeof(int));
      gridCombs = gC;
      hipMalloc((void**)&grid_gpu,gridCombs*nGridParams*sizeof(T));
      hipMemcpy(grid_gpu,grid_host,gridCombs*nGridParams*sizeof(T),hipMemcpyHostToDevice);
      
      sync_check("GridSearch: Copying Grid to GPU");

      // Set bounds
      bound_types_host = new int[NPARAMS];
      bounds_min_host = new float[NPARAMS];
      bounds_max_host = new float[NPARAMS];
      for(int p=0;p<NPARAMS;p++){
	bound_types_host[p]=bou_types[p];
	bounds_min_host[p]=bou_min[p];
	bounds_max_host[p]=bou_max[p];
      }
      hipMemcpyToSymbol(HIP_SYMBOL(GSbound_types),bound_types_host,NPARAMS*sizeof(int));
      hipMemcpyToSymbol(HIP_SYMBOL(GSbounds_min),bounds_min_host,NPARAMS*sizeof(float));
      hipMemcpyToSymbol(HIP_SYMBOL(GSbounds_max),bounds_max_host,NPARAMS*sizeof(float));
      sync_check("GridSearch: Setting Bounds");
   
      DEBUG=false;
      if(opts.debug.set()){
	DEBUG=true;
	debugVOX= atoi(opts.debug.value().data());
      }
    }
  }
  
  template <typename T>
  void GridSearch<T>::run(int nvox, int nmeas,
			  int CFP_size, int FixP_size,
			  T* meas, T* params,
			  T* CFP, T* FixP) 
  {
  
    long int amount_shared_mem = 0;
    amount_shared_mem += 2*VOXELS_BLOCK*sizeof(double); // cost function
    amount_shared_mem += (nmeas*CFP_size)*sizeof(T); // CFP
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // Parameters
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // trial_Params
    amount_shared_mem += VOXELS_BLOCK*sizeof(int); // boundsTest
        
    cout << "Shared Memory used in GridSearch kernel: " << amount_shared_mem << endl;
    
    int threads_block = VOXELS_BLOCK * THREADS_VOXEL;
    int nblocks=(nvox/VOXELS_BLOCK);
    if(nvox%VOXELS_BLOCK) nblocks++;
    
    if(!DEBUG){
      gridSearch_kernel<T,false><<<nblocks,threads_block,amount_shared_mem>>>(nGridParams,gridCombs,nmeas,CFP_size,FixP_size,meas,grid_gpu,params,CFP,FixP,debugVOX);
    }else{
      gridSearch_kernel<T,true><<<nblocks,threads_block,amount_shared_mem>>>(nGridParams,gridCombs,nmeas,CFP_size,FixP_size,meas,grid_gpu,params,CFP,FixP,debugVOX);
    }
    sync_check("GridSearch Kernel");
  }
  
  // Explicit Instantiations of the template
  template class GridSearch<float>;
  template class GridSearch<double>;
}


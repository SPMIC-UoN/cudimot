#include "hip/hip_runtime.h"
/* getPredictedSignal.cu

   Moises Hernandez-Fernandez - FMRIB Image Analysis Group
   
   Copyright (C) 2005 University of Oxford */

// Method to calculate the model predicted signal on the GPU of a group of voxels given the model parameters.

#include "getPredictedSignal.h"
#include "functions_gpu.h"
#include "modelparameters.h"
#include "macro_numerical.h"
#include "modelfunctions.h"

using namespace std;

namespace Cudimot{

#define VOXELS_BLOCK 8
#define THREADS_VOXEL 32 // Multiple of 32: Threads collaborating to compute a voxel. Do not change this, otherwise Synchronization will be needed and shuffles cannot be used
  
  template <typename T>
  __global__ void getPredictedSignal_kernel(
					    int nvox, // nvoxels
					    int nmeas, // nmeasurements
					    int nsamples,
					    int CFP_Tsize, //size*M-measurements
					    int FixP_Tsize, // fixed params: size*N-voxels
					    T* samples, // samples of estimated parameters 
					    T* CFP_global, // common fixed model parameters
					    T* FixP, // fixed model parameters
					    T* PredictedSignal)
  {
    // 1 block of threads process several voxels
    // Each warp processes 1 voxel
    int idVOX= (blockIdx.x*VOXELS_BLOCK)+int(threadIdx.x/THREADS_VOXEL);
    int idVOX_inBlock =  threadIdx.x/THREADS_VOXEL;
    int idSubVOX= threadIdx.x%THREADS_VOXEL;
    bool leader = (idSubVOX==0);  // Some steps are performed by only one thread of the warp
    
    ////////// DYNAMIC SHARED MEMORY ///////////
    extern __shared__ double shared[];	      	//Size:
    T* CFP = (T*)shared;		 	//nmeas*CMP_Tsize
    T* meanSamples = (T*) &CFP[nmeas*CFP_Tsize];//NPARAMS*VOXELS_BLOCK
    ////////////////////////////////////////////
    
    /// Copy common fixed model parameters to Shared Memory ///
    if(threadIdx.x==0){ // only one thread of the whole block. Common to all voxels
      for(int i=0;i<nmeas*CFP_Tsize;i++){
	CFP[i]=CFP_global[i];
      }
    }
    __syncthreads();
    ///////////////////////////////////////////////////////////
    
    ///////// each voxel/warp of the block points to its data///////////
    samples = &samples[idVOX*NPARAMS*nsamples]; // Global
    meanSamples = &meanSamples[idVOX_inBlock*NPARAMS];
    PredictedSignal = &PredictedSignal[idVOX*nmeas]; //Global
    FixP = &FixP[idVOX*FixP_Tsize]; // Global memory
    ////////////////////////////////////////////////////////////////////

    /// Ititialise shared values of each voxel: only the leader///
    if(leader){
      if(nsamples>1){
	for(int par=0;par<NPARAMS;par++){
	  T value=0;
	  for(int samp=0;samp<nsamples;samp++){
	    value+= samples[par*nsamples+samp];
	  }
	  meanSamples[par]=value/nsamples;
	}
      }else{
        #pragma unroll
	for(int par=0;par<NPARAMS;par++){
	  meanSamples[par]=samples[par];
	}
      }
    }
    __syncthreads();

    int idMeasurement=idSubVOX;
    int nmeas2compute = nmeas/THREADS_VOXEL;
    if (idSubVOX<(nmeas%THREADS_VOXEL)) nmeas2compute++;
    
    for(int iter=0;iter<nmeas2compute;iter++){
      T* myCFP = &CFP[idMeasurement*CFP_Tsize];
      T pred=Predicted_Signal(NPARAMS,meanSamples,myCFP,FixP);
      PredictedSignal[idMeasurement]=pred;
      idMeasurement+=THREADS_VOXEL;
    }
  }
  
  
  template <typename T>
  getPredictedSignal<T>::getPredictedSignal(){}
  
  template <typename T>
  void getPredictedSignal<T>::run(
				  int nvox, int nmeas, int nsamples,
				  int CFP_size, int FixP_size,
				  T* samples, T* CFP, T* FixP,
				  T* PredictedSignal) 
  {
    
    long int amount_shared_mem = 0;
    amount_shared_mem += (nmeas*CFP_size)*sizeof(T); // CFP
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); //mean_samples
    
    cout << "Shared Memory used in PredictedSignal kernel: " << amount_shared_mem << endl;
    
    int threads_block = VOXELS_BLOCK * THREADS_VOXEL;
    int nblocks=(nvox/VOXELS_BLOCK);
    if(nvox%VOXELS_BLOCK) nblocks++;
    
    getPredictedSignal_kernel<T><<<nblocks,threads_block,amount_shared_mem>>>(nvox,nmeas,nsamples,CFP_size,FixP_size,samples,CFP,FixP,PredictedSignal);
    sync_check("getPredictedSignal Kernel");
  }
  
  // Explicit Instantiations of the template
  template class getPredictedSignal<float>;
  template class getPredictedSignal<double>;
}

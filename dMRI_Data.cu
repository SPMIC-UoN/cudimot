#include "hip/hip_runtime.h"
/* dMRI_Data.cu

   Moises Hernandez-Fernandez - FMRIB Image Analysis Group
   
   Copyright (C) 2005 University of Oxford */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk

    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK


    LICENCE

    FMRIB Software Library, Release 6.0 (c) 2018, The University of
    Oxford (the "Software")

    The Software remains the property of the Oxford University Innovation
    ("the University").

    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.

    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.

    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.

    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    fsl@innovation.ox.ac.uk quoting Reference Project 9564, FSL.*/

#include "dMRI_Data.h"

namespace Cudimot{
  
  template <typename T>
  void dMRI_Data<T>::remove_NonPositive_entries(NEWMAT::ColumnVector& Voxdata){ 
    //Zero, Negative Entries can be obtained from spline interpolation 
    int pos; 
    float MinS=Voxdata.Minimum1(pos); 
    float MaxS=Voxdata.Maximum();
    if (MinS<=0 && MaxS>0){  
      //when there are some non-positive entries, but not all are zero
      vector<int> minpositions;
      while (MinS<=0){
	minpositions.push_back(pos);
	Voxdata(pos)=MaxS;    //temporarilly make the non-positive values Max
	MinS=Voxdata.Minimum1(pos);
      }
      MinS=Voxdata.Minimum(); //Now find the Minimum of positive entries
      for (unsigned int i=0; i<minpositions.size(); i++)
	Voxdata(minpositions[i])=MinS; //Replace non-positive entries with that minimum
    }
  }
  
  template <typename T>
  dMRI_Data<T>::dMRI_Data(){
    
    cudimotOptions& opts = cudimotOptions::getInstance();
    
    // Read the binary file with data (genereted previously in split_parts)
    ifstream in;
    long nbytes;
    string file_input;
    file_input.append(opts.partsdir.value());
    file_input.append("/part_");
    file_input.append(num2str(opts.idPart.value()));
    file_input.append("/data");
    
    in.open(file_input.data(), ios::in | ios::binary);
    in.read((char*)&nvox, 4);
    in.read((char*)&nmeas, 4);
    in.read((char*)&nbytes, sizeof(long));
    
    if(nvox<=0 || nmeas<=0){
      cerr << "CUDIMOT Error: The number of voxels and diffusion-weighted measurements in the input file must be greater than 0" << endl;
      exit (EXIT_FAILURE);
    }
    
    cout << "Number of Voxels to compute: " << nvox << endl;  
    cout << "Number of Measurements: " << nmeas << endl;  

    // Read diffusion-weighted measurements
    dataM.ReSize(nmeas,nvox);
    in.read((char*)&dataM(1,1),nbytes);
    in.close();
    
    // Data is divided into parts
    nparts=nvox/SIZE_PART;
    size_part=SIZE_PART;
    if(nvox%SIZE_PART) nparts++;
    size_last_part = nvox - ((nparts-1)*SIZE_PART);
    if(size_last_part<(SIZE_PART*0.5)){ 
      // if last part is too small, we distribute its voxels between the others parts
      if(nparts-1){ // More than 1 part
	size_part = size_part + size_last_part/(nparts-1);
	nparts--;
      }else{
	size_part = 0;
      }
      size_last_part = nvox - ((nparts-1)*size_part);
    }
    
    // Allocate memory on host and GPU for measurements
    int max_nvox =  max(size_part,size_last_part);
    // number of voxels can be a non-multiple of voxels per block, so somethreads could access to non-allocated memory. We use the closest upper multiple. The added voxels will be ignored.
    nvoxFit_part=int(max_nvox/MAX_VOXELS_BLOCK)*MAX_VOXELS_BLOCK;
    if(max_nvox%MAX_VOXELS_BLOCK) nvoxFit_part=nvoxFit_part+MAX_VOXELS_BLOCK;
    meas_host=new T[nvoxFit_part*nmeas];
    hipMalloc((void**)&meas_gpu,nvoxFit_part*nmeas*sizeof(T));
    sync_check("Allocating dMRI_Data on the GPU");
  }
  
  template <typename T>
  dMRI_Data<T>::~dMRI_Data(){
    //hipFree(meas_gpu);
    //sync_check("Deallocating dMRI_Data from GPU");
  }
  
  template <typename T>
  int dMRI_Data<T>::getNvoxFit_part() const{
    return nvoxFit_part;
  }
  
  template <typename T>
  int dMRI_Data<T>::getNmeas() const{
    return nmeas;
  }
  
  template <typename T>
  int dMRI_Data<T>::getNparts() const{
    return nparts;
  }
  
  // Returns size of part in the second parameter
  template <typename T>
  T* dMRI_Data<T>::getMeasPart(int part, int &sp){
    
    cudimotOptions& opts = cudimotOptions::getInstance();
    
    if(part>=nparts){
      cerr << "CUDIMOT Error: Trying to get an incorrect part of the data: " << part << ". There are only " << nparts << " parts and index starts at 0." << endl;
      exit(-1);
    }
    
    int size=size_part;
    int initial_vox=part*size_part;
    if(part==(nparts-1)){
      size=size_last_part;
    }
    
    cout << endl << endl << endl << "Part " << part+1 << " of " << nparts << ": processing " << size << " voxels" << endl;

    int vox=0;
    for(vox=0;vox<size;vox++){
      ColumnVector voxmeas;
      voxmeas=dataM.Column(initial_vox+vox+1);
      if(opts.rician.value()) remove_NonPositive_entries(voxmeas); //So that log(data) does not give infinity in the likelihood
      for(int m=0;m<nmeas;m++){
	meas_host[vox*nmeas+m]=voxmeas(m+1);
      }
    }
    // Fill with 0 the rest of the vector
    for(;vox<nvoxFit_part;vox++){
      for(int m=0;m<nmeas;m++){
	meas_host[vox*nmeas+m]=0;
      }
    }
    
    // Copy from host to GPU
    hipMemcpy(meas_gpu,meas_host,nvoxFit_part*nmeas*sizeof(T),hipMemcpyHostToDevice);
    sync_check("Copying dMRI_Data to GPU");
    sp=nvoxFit_part; 
    return meas_gpu;
  }
  
  template class dMRI_Data<float>;
  template class dMRI_Data<double>;
}

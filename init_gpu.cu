/*  init_gpu.cu
    
    Moises Hernandez-Fernandez FMRIB Image Analysis Group
    
    Copyright (C) 2005 University of Oxford  */

#include "checkcudacalls.h"
#include <fstream>

void init_gpu(){
  int *q;
  hipMalloc((void **)&q, sizeof(int));
  hipFree(q);
  sync_check("init_gpu");
  
  int device;
  hipGetDevice(&device);
  printf ("\n...................Allocated GPU %d...................\n", device);
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);
  sync_check("init_gpu");
} 

